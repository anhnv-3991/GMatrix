#include "hip/hip_runtime.h"
#include "NormalDistributionTransform.h"
#include "debug.h"

namespace gpu {
void GNormalDistributionTransform::computeTransform()
{
	nr_iterations_ = 0;
	converged_ = false;

	double gauss_c1, gauss_c2, gauss_d3;

	gauss_c1 = 10 * ( 1 - outlier_ratio_);
	gauss_c2 = outlier_ratio_ / pow(resolution_, 3);
	gauss_d3 = - log(gauss_c2);
	gauss_d1_ = -log(gauss_c1 + gauss_c2) - gauss_d3;
	gauss_d2_ = -2 * log((-log(gauss_c1 * exp(-0.5) + gauss_c2) - gauss_d3) / gauss_d1_);
}
}
